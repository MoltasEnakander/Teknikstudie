#include "beamformer.h"

#include <chrono>
#include <ctime>
#include <unistd.h>

#include <thread>

#include <iostream>
#include <fstream>

void free_resources(beamformingData* data)
{
    // free allocated memory
    free(data->beams);
    hipFree(data->gpu_beams);
    hipFree(data->a);
    hipFree(data->alpha);
    hipFree(data->b);
    hipFree(data->beta);
    fftwf_free(data->temp);
    fftwf_free(data->ordbuffer);
    fftwf_free(data->block);
    hipFree(data->gpu_block)
    hipFree(data->summedSignals);   
    fftwf_free(data->fft_data);    
    fftwf_free(data->filtered_data);    
    fftwf_free(data->LP_filter);        
    hipFree(data->summedSignals_fft);
    hipFree(data->summedSignals_fft_BP);
    hipFree(data->BP_filter);
    
    for (int i = 0; i < NUM_CHANNELS; ++i)
    {
        fftwf_destroy_plan(data->forw_plans[i]);
        fftwf_destroy_plan(data->back_plans[i]);                
    } 

    hipfftDestroy(data->planMany);    
    
    free(data);
}

// Checks the return value of a PortAudio function. Logs the message and exits
// if there was an error
static void checkErr(PaError err, beamformingData* data) {
    if (err != paNoError) {
        printf("PortAudio error: %s\n", Pa_GetErrorText(err));
        free_resources(data);
        exit(EXIT_FAILURE);
    }
}

// PortAudio stream callback function. Will be called after every
// BLOCK_LEN audio samples PortAudio captures. Used to process the
// resulting audio sample.
static int streamCallback(
    const void* inputBuffer, void* outputBuffer, unsigned long framesPerBuffer,
    const PaStreamCallbackTimeInfo* timeInfo, PaStreamCallbackFlags statusFlags,
    void* userData
)
{
    // Cast our input buffer to a float pointer (since our sample format is `pafloat32`)
    float* in = (float*)inputBuffer;

    beamformingData* data = (beamformingData*)userData;
    
    // keep track of when to stop listening
    int finished;
    unsigned long framesLeft = data->maxFrameIndex - data->frameIndex;

    if( framesLeft < framesPerBuffer )
    {
        data->frameIndex += framesLeft;
        finished = paComplete;
    }
    else
    {
        data->frameIndex += framesPerBuffer;
        finished = paContinue;
    }

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    
    for (int i = 0; i < NUM_CHANNELS; ++i) // sort the incoming buffer based on channel
    {       
        for (int j = 0; j < BLOCK_LEN; ++j)
        {            
            data->ordbuffer[i * BLOCK_LEN + j][0] = in[j * NUM_CHANNELS + i];
            //data->ordbuffer[i * BLOCK_LEN + j][0] = in[j];
            data->ordbuffer[i * BLOCK_LEN + j][1] = 0.0f;            
        }        
    }

    for (int i = 0; i < NUM_CHANNELS; ++i) // build data block to be processed
    {
        // 1. move the last part of the old input into the beginning of the block
        // 2. fill the rest of the block with BLOCK_LEN - TEMP values from the new input
        // 3. save the last TEMP values from the new input to the temp storage for use in next call
        std::memcpy(&(data->block[i * BLOCK_LEN]), &(data->temp[i * TEMP]), TEMP * sizeof(fftwf_complex)); 
        std::memcpy(&(data->block[i * BLOCK_LEN + TEMP]), &(data->ordbuffer[i * BLOCK_LEN]), (BLOCK_LEN - TEMP) * sizeof(fftwf_complex));
        std::memcpy(&(data->temp[i * TEMP]), &(data->ordbuffer[i * BLOCK_LEN + (BLOCK_LEN - TEMP)]), TEMP * sizeof(fftwf_complex));
    }

    for (int i = 0; i < NUM_CHANNELS; ++i) // calculate fft for each channel
    {
        fftwf_execute(data->forw_plans[i]);
    }

    // perform lowpass filtering in freq domain
    int resultID, dataID;
    for (int i = 0; i < NUM_CHANNELS; ++i) // for every channel
    {
        for (int j = 0; j < FFT_OUTPUT_SIZE; ++j) // for all samples
        {                
            // j denotes frequency bin            
            // i denotes the channel
            resultID = j + i * FFT_OUTPUT_SIZE;
            dataID = j + i * FFT_OUTPUT_SIZE;
            data->filtered_data[resultID][0] = data->fft_data[dataID][0] * data->LP_filter[j][0] - data->fft_data[dataID][1] * data->LP_filter[j][1];
            data->filtered_data[resultID][1] = data->fft_data[dataID][0] * data->LP_filter[j][1] + data->fft_data[dataID][1] * data->LP_filter[j][0];                
        }
        // inverse fourier transform to get back signals in time domain.        
        fftwf_execute(data->back_plans[i]); // amplitude gain BLOCK_LEN
    }

    // copy data blocks to gpu
    hipMemcpy(data->gpu_block, data->block, BLOCK_LEN*NUM_CHANNELS*sizeof(fftwf_complex), hipMemcpyHostToDevice); // copy buffer to GPU memory    

    // create beams    
    beamforming<<<data->numBlocks, data->threadsPerBlock>>>(data->gpu_block, data->a, data->b, data->alpha, data->beta, data->summedSignals);
    hipDeviceSynchronize();

    hipfftExecC2C(data->planMany, data->summedSignals, data->summedSignals_fft, HIPFFT_FORWARD);
    hipDeviceSynchronize();

    bandpass_filtering<<<data->numBlocks, data->threadsPerBlock>>>(data->summedSignals_fft_BP, data->summedSignals_fft, data->BP_filter, data->gpu_beams);
    hipDeviceSynchronize();    

    // copy the intensity of the beams to the cpu
    hipMemcpy(data->beams, data->gpu_beams, NUM_BEAMS*NUM_BEAMS*NUM_FILTERS*sizeof(float), hipMemcpyDeviceToHost);
    
    end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed = end-start;

    std::cout << "elapsed: " << elapsed.count() << "s\n";

    return finished;
}

int main() 
{
    // Initialize PortAudio
    PaError err;
    err = Pa_Initialize();
    checkErr(err, nullptr);

    // --------------------------------------------------------------------------------------------------------------
    // ------------------------ List all available audio devices and look for desired device ------------------------
    // --------------------------------------------------------------------------------------------------------------
    int numDevices = Pa_GetDeviceCount();
    printf("Number of devices: %d\n", numDevices);

    if (numDevices < 0){
        printf("Error getting device count.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }
    else if (numDevices == 0){
        printf("There are no available audio devices on this machine.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }

    int device = -1;
    const PaDeviceInfo* deviceInfo;
    for (int i = 0; i < numDevices; i++)
    {
        deviceInfo = Pa_GetDeviceInfo(i);
        printf("Device %d:\n", i);
        printf("    name: %s\n", deviceInfo->name);
        printf("    maxInputChannels: %d\n", deviceInfo->maxInputChannels);
        printf("    maxOutputChannels: %d\n", deviceInfo->maxOutputChannels);
        printf("    defaultSampleRate: %f\n", deviceInfo->defaultSampleRate);

        if (strcmp(deviceInfo->name, DEVICE_NAME) == 0)
        {
            device = i;
        }
    }

    if (device == -1){
        printf("\nDevice: %s not found!\n", DEVICE_NAME);
        Pa_Terminate();
        exit(EXIT_FAILURE);
    }

    printf("Device = %d\n", device);
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------    

    // setup necessary data containers for the beamforming
    beamformingData* data = (beamformingData*)malloc(sizeof(beamformingData));
    data->maxFrameIndex = NUM_SECONDS * SAMPLE_RATE; // Record for a few seconds.
    data->frameIndex = 0;

    if (NUM_BEAMS * NUM_BEAMS > MAX_THREADS_PER_BLOCK){
        data->numBlocks = (NUM_BEAMS * NUM_BEAMS) % MAX_THREADS_PER_BLOCK + 1;
        data->threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
    }
    else{
        data->numBlocks = 1;
        data->threadsPerBlock = dim3(NUM_BEAMS * NUM_BEAMS);
    }

    printf("Setting up fir filters.\n");    
    py::scoped_interpreter python;

    py::function my_func =
        py::reinterpret_borrow<py::function>(
            py::module::import("beamformer.filtercreation").attr("filtercreation")
    );    
    
    py::list res = my_func(NUM_FILTERS, NUM_TAPS, BANDWIDTH); // create the filters
    // temporary save state of data
    std::vector<float> taps;
    for (py::handle obj : res) {  // iterators!
        taps.push_back(obj.attr("__float__")().cast<float>());
    }

    py::list res2 = my_func(1, NUM_TAPS, 10000.0 / 22050.0);
    // temporary save state of data
    std::vector<float> taps2;
    for (py::handle obj : res2) {  // iterators!
        taps2.push_back(obj.attr("__float__")().cast<float>());
    }

    // transfer data for real, goal is to get a buffer that looks like (with zero-padded signals):
    // filter1[0], filter1[1], ..., 0, 0, 0, filter2[0], filter2[1], ..., 0, 0, 0
    // -------- BLOCK_LEN samples ---------, -------- BLOCK_LEN samples --------- 
    fftwf_complex* firfilters = (fftwf_complex*)malloc(BLOCK_LEN * NUM_FILTERS * sizeof(fftwf_complex));
    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        for (int j = 0; j < BLOCK_LEN; ++j)
        {
            if (j < NUM_TAPS)
                firfilters[i * BLOCK_LEN + j][0] = taps[NUM_TAPS * i + j];
            else
                firfilters[i * BLOCK_LEN + j][0] = 0.0; // zero pad filters
            firfilters[i * BLOCK_LEN + j][1] = 0.0;
        }
    }
    taps.clear();

    fftwf_complex* lpfilter = (fftwf_complex*)malloc(BLOCK_LEN * sizeof(fftwf_complex));
    for (int i = 0; i < BLOCK_LEN; ++i)
    {
        if (i < NUM_TAPS)
            lpfilter[i][0] = taps2[i];
        else
            lpfilter[i][0] = 0.0; // zero pad filters
        lpfilter[i][1] = 0.0;
    }
    taps2.clear();

    // apply fft to filters
    fftwf_complex* firfiltersfft = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_FILTERS * sizeof(fftwf_complex));
    data->LP_filter = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * sizeof(fftwf_complex));
    fftwf_plan filter_plans[NUM_FILTERS];
    fftwf_plan lp_filter_plan;
    for (int i = 0; i < NUM_FILTERS; ++i) // create the plans for calculating the fft of each filter block
    {
        filter_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &firfilters[i * BLOCK_LEN], &firfiltersfft[i * FFT_OUTPUT_SIZE], FFTW_FORWARD, FFTW_ESTIMATE);
    }
    lp_filter_plan = fftwf_plan_dft_1d(BLOCK_LEN, lpfilter, data->LP_filter, FFTW_FORWARD, FFTW_ESTIMATE);

    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        fftwf_execute(filter_plans[i]);
    }
    fftwf_execute(lp_filter_plan);
    
    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        fftwf_destroy_plan(filter_plans[i]);
    }
    fftwf_destroy_plan(lp_filter_plan);

    hipMalloc(&(data->BP_filter), sizeof(hipfftComplex) * BLOCK_LEN * NUM_FILTERS);
    hipMemcpy(data->BP_filter, firfiltersfft, sizeof(hipfftComplex) * BLOCK_LEN * NUM_FILTERS, hipMemcpyHostToDevice);    

    free(firfilters);
    free(firfiltersfft);
    free(lpfilter);    

    printf("Create interpolation data.\n");
    float* theta = linspace(MIN_VIEW, NUM_BEAMS);
    float* phi = linspace(MIN_VIEW, NUM_BEAMS);
    float* delay = calcDelays(theta, phi);

    int* a = calca(delay);
    int* b = calcb(a);
    float* alpha = calcalpha(delay, b);
    float* beta = calcbeta(alpha);

    hipMalloc(&(data->a), sizeof(int) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->b), sizeof(int) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->alpha), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->beta), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMemcpy(data->a, a, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->b, b, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->alpha, alpha, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->beta, beta, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);

    free(theta); free(phi); free(delay); free(a); free(b); free(alpha); free(beta); // free memory which does not have to be allocated anymore*/    

    printf("Create remaining buffers\n");
    data->beams = (float*)malloc(NUM_BEAMS * NUM_BEAMS * NUM_FILTERS * sizeof(float));
    std::memset(data->beams, 0.0, NUM_BEAMS * NUM_BEAMS * NUM_FILTERS * sizeof(float));
    hipMalloc(&(data->gpu_beams), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_FILTERS);

    hipMalloc(&(data->gpu_block), sizeof(hipfftComplex) * NUM_CHANNELS * BLOCK_LEN);

    data->temp = (fftwf_complex*)fftwf_malloc(TEMP * NUM_CHANNELS * sizeof(fftwf_complex));
    for (int i = 0; i < TEMP * NUM_CHANNELS; ++i)
    {
        data->temp[i][0] = 0.0;
        data->temp[i][1] = 0.0;
    }

    data->ordbuffer = (fftwf_complex*)fftwf_malloc(BLOCK_LEN * NUM_CHANNELS * sizeof(fftwf_complex));
    data->block = (fftwf_complex*)fftwf_malloc(BLOCK_LEN * NUM_CHANNELS * sizeof(fftwf_complex));
    
    hipMalloc(&(data->summedSignals), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN);
    hipMalloc(&(data->summedSignals_fft), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN);
    hipMalloc(&(data->summedSignals_fft_BP), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN * NUM_FILTERS);

    data->fft_data = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_CHANNELS * sizeof(fftwf_complex));
    data->filtered_data = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_CHANNELS * sizeof(fftwf_complex));    

    for (int i = 0; i < BLOCK_LEN * NUM_CHANNELS; ++i)
    {
        data->ordbuffer[i][0] = 0.0;
        data->ordbuffer[i][1] = 0.0;
    }
    
    for (int i = 0; i < BLOCK_LEN * NUM_CHANNELS; ++i)
    {
        data->block[i][0] = 0.0;
        data->block[i][1] = 0.0;
    }

    printf("Creating fft plans.\n");
    int n[1] = {BLOCK_LEN};
    int inembed[] = {BLOCK_LEN};
    int onembed[] = {BLOCK_LEN};
    
    hipfftPlanMany(&(data->planMany), 1, n, inembed, 1, BLOCK_LEN, onembed, 1, BLOCK_LEN, HIPFFT_C2C, NUM_BEAMS*NUM_BEAMS);

    for (int i = 0; i < NUM_CHANNELS; ++i) // create the plans for calculating the fft of each channel block
    {
        data->forw_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &data->block[i * BLOCK_LEN], &data->fft_data[i * FFT_OUTPUT_SIZE], FFTW_FORWARD, FFTW_ESTIMATE); // NUM_CHANNELS channels for each block which requires FFT_OUTPUT_SIZE spots to store the fft data
        data->back_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &data->filtered_data[i * FFT_OUTPUT_SIZE], &data->block[i * BLOCK_LEN], FFTW_BACKWARD, FFTW_ESTIMATE);
    }
    
    printf("Defining stream parameters.\n");
    PaStreamParameters inputParameters;
    memset(&inputParameters, 0, sizeof(inputParameters));
    inputParameters.channelCount = NUM_CHANNELS;
    inputParameters.device = device;
    inputParameters.hostApiSpecificStreamInfo = NULL;
    inputParameters.sampleFormat = paFloat32;
    inputParameters.suggestedLatency = Pa_GetDeviceInfo(device)->defaultLowInputLatency;

    // Open the PortAudio stream
    printf("Starting stream.\n");    
    PaStream* stream;
    err = Pa_OpenStream(
        &stream,
        &inputParameters,
        NULL,
        SAMPLE_RATE,
        BLOCK_LEN,
        paNoFlag,
        streamCallback,
        data
    );
    checkErr(err, data);

    // Begin capturing audio
    err = Pa_StartStream(stream);
    checkErr(err, data);

    FILE* signal = popen("gnuplot", "w");
    //FILE* signal2 = popen("gnuplot", "w");
    //FILE* signal3 = popen("gnuplot", "w");

    while( ( err = Pa_IsStreamActive( stream ) ) == 1 )    
    {
        // plot beamforming results in color map
        fprintf(signal, "unset key\n");
        fprintf(signal, "set pm3d\n");
        fprintf(signal, "set view map\n");
        fprintf(signal, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal, "plot '-' matrix with image\n");
        
        for(int i = 3 * NUM_BEAMS * NUM_BEAMS; i < 4 * NUM_BEAMS * NUM_BEAMS; i++) // plot map for the lowest frequency band    
        {
            fprintf(signal, "%f ", data->beams[i]);            
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal, "\n");            
        }
        
        fprintf(signal, "\ne\n");

        fflush(signal);

        /*fprintf(signal2, "unset key\n");
        fprintf(signal2, "set pm3d\n");
        fprintf(signal2, "set view map\n");
        fprintf(signal2, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal2, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal2, "plot '-' matrix with image\n");
        
        for(int i = 1 * NUM_BEAMS * NUM_BEAMS; i < 2 * NUM_BEAMS * NUM_BEAMS; i++)
        {
            fprintf(signal2, "%f ", data->beams[i]);
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal2, "\n");            
        }
        
        fprintf(signal2, "\ne\n");        
        fflush(signal2);

        fprintf(signal3, "unset key\n");
        fprintf(signal3, "set pm3d\n");
        fprintf(signal3, "set view map\n");
        fprintf(signal3, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal3, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal3, "plot '-' matrix with image\n");
        
        for(int i = 2 * NUM_BEAMS * NUM_BEAMS; i < 3 * NUM_BEAMS * NUM_BEAMS; i++)
        {
            fprintf(signal3, "%f ", data->beams[i]);            
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal3, "\n");            
        }
        
        fprintf(signal3, "\ne\n");        
        fflush(signal3);*/

        // Display the buffered changes to stdout in the terminal
        //fflush(stdout);
    }    

    // Stop capturing audio
    err = Pa_StopStream(stream);
    checkErr(err, data);

    // Close the PortAudio stream
    err = Pa_CloseStream(stream);
    checkErr(err, data);

    // Terminate PortAudio
    err = Pa_Terminate();
    checkErr(err, data);

    free_resources(data);
    return 0;
}