#include "hip/hip_runtime.h"
#include "gpu_beamformer.h"

#include "matplotlibcpp.h"
namespace plt = matplotlibcpp;

#include <chrono>
#include <ctime>
#include <unistd.h>

__global__
void interpolateChannels(const float* inputBuffer, float* summedSignals, const int i, const int* a, const int* b, const float* alpha, const float* beta)
{
    int id;
    int l1 = blockIdx.x * blockDim.x + threadIdx.x; // internal index of this thread
    int l2 = blockIdx.x * blockDim.x + threadIdx.x + i * FRAMES_PER_BUFFER; // global index of this thread
    for (int k = 0; k < NUM_CHANNELS; ++k)
    {
        id = k + i * NUM_CHANNELS;        
        if (max(0, -a[id]) == 0 && l1 < FRAMES_PER_BUFFER - a[id]) // a >= 0
            summedSignals[l2] += alpha[id] * inputBuffer[(l1+a[id])*NUM_CHANNELS + k]; // do not write to the a[id] end positions
        else if (max(0, -a[id]) > 0 && l1 >= a[id]) 
            summedSignals[l2] += alpha[id] * inputBuffer[(l1+a[id])*NUM_CHANNELS + k]; // do not write to the first a[id]-1 positions

        if (max(0, -b[id]) == 0 && l1 < FRAMES_PER_BUFFER - b[id]) // b >= 0
            summedSignals[l2] += beta[id] * inputBuffer[(l1+b[id])*NUM_CHANNELS + k]; // do not write to the b[id] end positions
        else if (max(0, -b[id]) > 0 && l1 >= b[id]) 
            summedSignals[l2] += beta[id] * inputBuffer[(l1+b[id])*NUM_CHANNELS + k]; // do not write to the first b[id]-1 positions        
    }
}

__global__ 
void beamforming(const float* inputBuffer, float* beams, const int* a, const int* b, const float* alpha, const float* beta, float* summedSignals)
{    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= NUM_VIEWS * NUM_VIEWS){
        return;
    }

    // interpolate channels    
    interpolateChannels<<<(FRAMES_PER_BUFFER+255)/256, 256>>>(inputBuffer, summedSignals, i, a, b, alpha, beta);
    hipDeviceSynchronize();

    int idx;
    float beamstrength = 0.0f;
    // normalize
    for (int q = 0; q < FRAMES_PER_BUFFER; ++q)
    {
        idx = q + i * FRAMES_PER_BUFFER;
        summedSignals[idx] /= NUM_CHANNELS;
        summedSignals[idx] = summedSignals[idx] * summedSignals[idx] / FRAMES_PER_BUFFER;
        beamstrength += summedSignals[idx];
    }

    beams[i] = 10 * log10(beamstrength);
}

// Checks the return value of a PortAudio function. Logs the message and exits
// if there was an error
static void checkErr(PaError err) {
    if (err != paNoError) {
        printf("PortAudio error: %s\n", Pa_GetErrorText(err));
        exit(EXIT_FAILURE);
    }
}

// PortAudio stream callback function. Will be called after every
// `2*FRAMES_PER_BUFFER` audio samples PortAudio captures. Used to process the
// resulting audio sample.
static int streamCallback(
    const void* inputBuffer, void* outputBuffer, unsigned long framesPerBuffer, // framesPerBuffer = 2 * FRAMES_PER_BUFFER
    const PaStreamCallbackTimeInfo* timeInfo, PaStreamCallbackFlags statusFlags,
    void* userData
) {
    // Cast our input buffer to a float pointer (since our sample format is `paFloat32`)
    float* in = (float*)inputBuffer;

    // We will not be modifying the output buffer. This line is a no-op.
    (void)outputBuffer;

    beamformingData* data = (beamformingData*)userData;
    
    int finished;
    unsigned long framesLeft = data->maxFrameIndex - data->frameIndex;

    if( framesLeft < framesPerBuffer )
    {
        data->frameIndex += framesLeft;
        finished = paComplete;
    }
    else
    {
        data->frameIndex += framesPerBuffer;
        finished = paContinue;
    }

    hipMemcpy(data->buffer, in, FRAMES_PER_BUFFER*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice); // copy buffer to GPU memory   
    
    // beamform
    int numBlocks;
    dim3 threadsPerBlock;
    if (NUM_VIEWS * NUM_VIEWS > MAX_THREADS_PER_BLOCK){
        numBlocks = (NUM_VIEWS * NUM_VIEWS) % MAX_THREADS_PER_BLOCK + 1;
        threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
    }
    else{
        numBlocks = 1;
        threadsPerBlock = dim3(NUM_VIEWS * NUM_VIEWS);
    }
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    beamforming<<<numBlocks, threadsPerBlock>>>(data->buffer, data->gpubeams, data->a, data->b, data->alpha, data->beta, data->summedSignals);
    hipDeviceSynchronize();
    end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed = end-start;

    std::cout << "elapsed: " << elapsed.count() << "s\n";

    hipMemcpy(data->cpubeams, data->gpubeams, NUM_VIEWS*NUM_VIEWS*sizeof(float), hipMemcpyDeviceToHost);

    int maxID = 0;
    float maxVal = data->cpubeams[0];

    for (int i = 1; i < NUM_VIEWS * NUM_VIEWS; i++)
    {
        if (maxVal < data->cpubeams[i]){
            maxID = i;
            maxVal = data->cpubeams[i];
        }        
    }

    // convert 1d index to 2d index
    data->thetaID = maxID % int(NUM_VIEWS);
    data->phiID = maxID / int(NUM_VIEWS);

    return finished;
}

void listen_live() 
{
    // Initialize PortAudio
    PaError err;
    err = Pa_Initialize();
    checkErr(err);

    // --------------------------------------------------------------------------------------------------------------
    // ------------------------ List all available audio devices and look for desired device ------------------------
    // --------------------------------------------------------------------------------------------------------------
    int numDevices = Pa_GetDeviceCount();
    printf("Number of devices: %d\n", numDevices);

    if (numDevices < 0){
        printf("Error getting device count.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }
    else if (numDevices == 0){
        printf("There are no available audio devices on this machine.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }

    int device = -1;
    const PaDeviceInfo* deviceInfo;
    for (int i = 0; i < numDevices; i++)
    {
        deviceInfo = Pa_GetDeviceInfo(i);
        printf("Device %d:\n", i);
        printf("    name: %s\n", deviceInfo->name);
        printf("    maxInputChannels: %d\n", deviceInfo->maxInputChannels);
        printf("    maxOutputChannels: %d\n", deviceInfo->maxOutputChannels);
        printf("    defaultSampleRate: %f\n", deviceInfo->defaultSampleRate);

        if (strcmp(deviceInfo->name, DEVICE_NAME) == 0)
        {
            device = i;
        }
    }

    if (device == -1){
        printf("\nDevice: %s not found!\n", DEVICE_NAME);
        Pa_Terminate();
        exit(EXIT_FAILURE);
    }

    printf("Device = %d\n", device);
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------

    // setup interpolation data for the views and channels
    theta = linspace(MIN_VIEW, NUM_VIEWS);
    phi = linspace(MIN_VIEW, NUM_VIEWS);
    delay = calcDelays();
    a = calca();
    b = calcb();
    alpha = calcalpha();
    beta = calcbeta();

    // Define stream capture specifications
    PaStreamParameters inputParameters;
    memset(&inputParameters, 0, sizeof(inputParameters));
    inputParameters.channelCount = NUM_CHANNELS;
    inputParameters.device = device;
    inputParameters.hostApiSpecificStreamInfo = NULL;
    inputParameters.sampleFormat = paFloat32;
    inputParameters.suggestedLatency = Pa_GetDeviceInfo(device)->defaultLowInputLatency;

    beamformingData* data = (beamformingData*)malloc(sizeof(beamformingData));
    data->maxFrameIndex = NUM_SECONDS * SAMPLE_RATE; // Record for a few seconds.
    data->frameIndex = 0;
    
    hipMalloc(&(data->buffer), sizeof(float) * FRAMES_PER_BUFFER * NUM_CHANNELS);    
    hipMalloc(&(data->gpubeams), sizeof(float) * NUM_VIEWS * NUM_VIEWS);
    hipMalloc(&(data->a), sizeof(int) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->b), sizeof(int) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->alpha), sizeof(float) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->beta), sizeof(float) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->summedSignals), sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES_PER_BUFFER);    
    
    hipMemcpy(data->a, a, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->b, b, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->alpha, alpha, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->beta, beta, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    
    data->cpubeams = (float*)malloc(NUM_VIEWS*NUM_VIEWS*sizeof(float));

    // Open the PortAudio stream
    PaStream* stream;
    err = Pa_OpenStream(
        &stream,
        &inputParameters,
        NULL,
        SAMPLE_RATE,
        FRAMES_PER_BUFFER,
        paNoFlag,
        streamCallback,
        data
    );
    checkErr(err);

    // Begin capturing audio
    err = Pa_StartStream(stream);
    checkErr(err);

    FILE* signal = popen("gnuplot", "w");    
    //const int nrows = 4, ncols = 4;    
    //int row, col;

    while( ( err = Pa_IsStreamActive( stream ) ) == 1 )
    {
        //Pa_Sleep(100);
        // plot maximum direction
        plt::figure(1);
        plt::title("Max direction plot");
        plt::clf();
        plt::scatter(std::vector<float>{theta[data->thetaID] * 180.0f / (float)M_PI}, std::vector<float>{phi[data->phiID] * 180.0f / (float)M_PI}, 25.0, {{"color", "red"}});
        plt::xlim(MIN_VIEW, MAX_VIEW);
        plt::ylim(MIN_VIEW, MAX_VIEW);
        plt::xlabel("theta");
        plt::ylabel("phi");
        plt::grid(true);
        plt::pause(0.15);
        //printf("theta = %f\n", data->theta );
        //printf("phi = %f\n", data->phi );
        //printf("maxframeindex = %d\n", data->maxFrameIndex );
        //printf("frameindex = %d\n", data->frameIndex );
        //fflush(stdout);

        // plot frequency contents of channels
        /*plt::figure(2);
        plt::title("Frequency contents");
        plt::clf();
        for(int w = 0; w < NUM_CHANNELS; ++w){
            row = w / 4;
            col = w % 4;
            plt::subplot2grid(nrows, ncols, row, col);
            plt::plot({1.0,2.0,3.0,4.0});
            plt::xlabel("freq bin");
        }
        //plt::show();
        plt::pause(0.02);*/
        


        // plot beamforming results in color map
        fprintf(signal, "unset key\n");
        fprintf(signal, "set pm3d\n");
        fprintf(signal, "set view map\n");
        fprintf(signal, "set xrange [ -0.5 : %f ] \n", NUM_VIEWS-0.5);
        fprintf(signal, "set yrange [ -0.5 : %f ] \n", NUM_VIEWS-0.5);
        fprintf(signal, "plot '-' matrix with image\n");
        
        for(int i = 0; i < NUM_VIEWS * NUM_VIEWS; ++i)    
        {
            fprintf(signal, "%f ", data->cpubeams[i]);
            if ((i+1) % NUM_VIEWS == 0)
                fprintf(signal, "\n");
        }
        
        fprintf(signal, "e\n");
        fprintf(signal, "e\n");
        fflush(signal);    

        // Display the buffered changes to stdout in the terminal
        fflush(stdout);
    }    

    // Stop capturing audio
    err = Pa_StopStream(stream);
    checkErr(err);

    // Close the PortAudio stream
    err = Pa_CloseStream(stream);
    checkErr(err);

    // Terminate PortAudio
    err = Pa_Terminate();
    checkErr(err);

    // free allocated memory
    hipFree(data->buffer);
    hipFree(data->gpubeams);    
    hipFree(data->a);
    hipFree(data->b);
    hipFree(data->alpha);
    hipFree(data->beta);    
    free(delay);
    free(theta);
    free(phi);    
    free(a);
    free(b);
    free(alpha);
    free(beta);
    free(data->cpubeams);
    free(data);

    printf("\n");    

    //return EXIT_SUCCESS;
}

void beamform_prerecorded(unsigned long framesPerBuffer, beamformingData* data) 
{
    unsigned long framesLeft = data->maxFrameIndex - data->frameIndex;

    printf("Frames left: %d\n", framesLeft);

    //int frame = data->frameIndex;

    if( framesLeft < framesPerBuffer )
    {
        data->frameIndex += framesLeft;        
    }
    else
    {
        data->frameIndex += framesPerBuffer;        
    }   

    // beamform
    int numBlocks;
    dim3 threadsPerBlock;
    if (NUM_VIEWS * NUM_VIEWS > MAX_THREADS_PER_BLOCK){
        numBlocks = (NUM_VIEWS * NUM_VIEWS) % MAX_THREADS_PER_BLOCK + 1;
        threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
    }
    else{
        numBlocks = 1;
        threadsPerBlock = dim3(NUM_VIEWS * NUM_VIEWS);
    }
    beamforming<<<numBlocks, threadsPerBlock>>>(data->buffer, data->gpubeams, data->a, data->b, data->alpha, data->beta, data->summedSignals);
    hipDeviceSynchronize();

    hipMemcpy(data->cpubeams, data->gpubeams, NUM_VIEWS*NUM_VIEWS*sizeof(float), hipMemcpyDeviceToHost);

    int maxID = 0;
    float maxVal = data->cpubeams[0];

    for (int i = 1; i < NUM_VIEWS * NUM_VIEWS; i++)
    {
        if (maxVal < data->cpubeams[i]){
            maxID = i;
            maxVal = data->cpubeams[i];
        }        
    }

    // convert 1d index to 2d index
    data->thetaID = maxID % int(NUM_VIEWS);
    data->phiID = maxID / int(NUM_VIEWS);    
}


void listen_prerecorded(std::vector<AudioFile<float>>& files)
{
    int length = files[0].getNumSamplesPerChannel() * NUM_CHANNELS;
    int q = 1;
    while (length == 0 && q < NUM_CHANNELS){ // some channels may be faulty and have 0 samples, make sure that length is longer than 0
        printf("length: %d\n", length);
        length = files[q].getNumSamplesPerChannel() * NUM_CHANNELS;
        q++;
    }
    assert(length > 0); // if all channels are 0 samples long this will alert

    float* inputBuffer = (float*)malloc(length*sizeof(float));
    float* cpyinputBuffer = inputBuffer;

    // build the inputbuffer
    int idx = 0;
    int idx2 = 0;
    int channel = 0; // channel zero of each file, since each file is mono
    for (int i = 0; i < length; ++i)
    {
        idx = i % NUM_CHANNELS;
        idx2 = i / NUM_CHANNELS;
        if (files[idx].getNumSamplesPerChannel() > 0) // if sample exist, copy it
            inputBuffer[i] = files[idx].samples[channel][idx2];
        else
            inputBuffer[i] = 0; // if the channel does not have any samples, fill with zero
    }

    theta = linspace(MIN_VIEW, NUM_VIEWS);
    phi = linspace(MIN_VIEW, NUM_VIEWS);
    delay = calcDelays();
    a = calca();
    b = calcb();
    alpha = calcalpha();
    beta = calcbeta();

    beamformingData* data = (beamformingData*)malloc(sizeof(beamformingData));
    data->maxFrameIndex = files[0].getNumSamplesPerChannel();
    data->frameIndex = 0;

    hipMalloc(&(data->buffer), sizeof(float) * length);
    hipMalloc(&(data->gpubeams), sizeof(float) * NUM_VIEWS * NUM_VIEWS);
    hipMalloc(&(data->a), sizeof(int) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->b), sizeof(int) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->alpha), sizeof(float) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->beta), sizeof(float) * NUM_VIEWS * NUM_VIEWS * NUM_CHANNELS);
    hipMalloc(&(data->summedSignals), sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES_PER_BUFFER);    
    
    hipMemcpy(data->buffer, cpyinputBuffer, length*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->a, a, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->b, b, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->alpha, alpha, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->beta, beta, NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    
    data->cpubeams = (float*)malloc(NUM_VIEWS*NUM_VIEWS*sizeof(float));

    double duration = (double)FRAMES_PER_BUFFER / (double)files[0].getSampleRate();
    printf("duration: %f %d %d\n", duration, FRAMES_PER_BUFFER, files[0].getSampleRate());
    assert(duration > 0); // TODO: make robust

    FILE* signal = popen("gnuplot", "w");
    std::chrono::time_point<std::chrono::system_clock> start, end;
    while (data->frameIndex < data->maxFrameIndex)
    {        
        start = std::chrono::system_clock::now();
        // do calculations and drawings

        beamform_prerecorded(FRAMES_PER_BUFFER, data);
        // the entire buffer is already created, but this simulates the stream, after the first FRAMES_PER_BUFFER has been processed, update the pointer to point 
        // to the next frames that should be processed by the beamforming algorithm
        data->buffer += FRAMES_PER_BUFFER*NUM_CHANNELS; 

        plt::figure(1);
        plt::title("Max direction plot");
        plt::clf();
        plt::scatter(std::vector<float>{theta[data->thetaID] * 180.0f / (float)M_PI}, std::vector<float>{phi[data->phiID] * 180.0f / (float)M_PI}, 25.0, {{"color", "red"}});
        plt::xlim(MIN_VIEW, MAX_VIEW);
        plt::ylim(MIN_VIEW, MAX_VIEW);
        plt::xlabel("theta");
        plt::ylabel("phi");
        plt::grid(true);
        plt::pause(0.15);

        // plot beamforming results in color map
        fprintf(signal, "unset key\n");
        fprintf(signal, "set pm3d\n");
        fprintf(signal, "set view map\n");
        fprintf(signal, "set xrange [ -0.5 : %f ] \n", NUM_VIEWS-0.5);
        fprintf(signal, "set yrange [ -0.5 : %f ] \n", NUM_VIEWS-0.5);
        fprintf(signal, "plot '-' matrix with image\n");
        
        for(int i = 0; i < NUM_VIEWS * NUM_VIEWS; ++i)    
        {
            fprintf(signal, "%f ", data->cpubeams[i]);
            if ((i+1) % NUM_VIEWS == 0)
                fprintf(signal, "\n");
        }
        
        fprintf(signal, "e\n");
        fprintf(signal, "e\n");
        fflush(signal);    

        // Display the buffered changes to stdout in the terminal
        fflush(stdout);

        end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed = end-start;

        std::cout << "elapsed: " << elapsed.count() << "s\n";

        //sleep(duration - elapsed.count()); // sleep for some time so that the playback "appears" like real time
    }

    // free allocated memory
    free(inputBuffer);
    free(delay);
    free(theta);
    free(phi);    
    free(a);
    free(b);
    free(alpha);
    free(beta);
    free(data->cpubeams);
    free(data);
    hipFree(data->buffer);
    hipFree(data->gpubeams);    
    hipFree(data->a);
    hipFree(data->b);
    hipFree(data->alpha);
    hipFree(data->beta);
}

/////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////
/////////////////// UTILITY FUNCTIONS ///////////////////////
/////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////

float* linspace(int a, int num)
{
    // create a vector of length num
    //std::vector<double> v(NUM_VIEWS, 0);    
    float* f = (float*)malloc(NUM_VIEWS*sizeof(float));
             
    // now assign the values to the vector
    for (int i = 0; i < num; i++)
    {
        f[i] = (a + i * VIEW_INTERVAL) * M_PI / 180.0f;
    }
    return f;
}

float* calcDelays()
{
    float* d = (float*)malloc(NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float));   

    int pid = 0;
    int tid = 0;
    for (int i = 0; i < NUM_VIEWS * NUM_VIEWS; ++i){
        for (int k = 0; k < NUM_CHANNELS; ++k){
            d[k + i * NUM_CHANNELS] = -(ya[k] * sinf(theta[tid]) * cosf(phi[pid]) + za[k] * sinf(phi[pid])) * ARRAY_DIST / C * SAMPLE_RATE;
        }
        tid++;
        if (tid >= NUM_VIEWS){
            tid = 0;
            pid++;
        }
    }
    return d;
}

int* calca()
{
    int* a = (int*)malloc(NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int));
    for (int i = 0; i < NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS; ++i)
    {
        a[i] = floor(delay[i]);
    }
    return a;
}

int* calcb()
{
    int* b = (int*)malloc(NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(int));
    for (int i = 0; i < NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS; ++i)
    {
        b[i] = a[i] + 1;
    }
    return b;
}

float* calcalpha()
{
    float* alpha = (float*)malloc(NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float));
    for (int i = 0; i < NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS; ++i)
    {
        alpha[i] = b[i] - delay[i];
    }
    return alpha;
}

float* calcbeta()
{
    float* beta = (float*)malloc(NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS*sizeof(float));
    for (int i = 0; i < NUM_VIEWS*NUM_VIEWS*NUM_CHANNELS; ++i)
    {
        beta[i] = 1 - alpha[i];
    }
    return beta;
}