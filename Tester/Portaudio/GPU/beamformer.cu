#include "hip/hip_runtime.h"
#include "beamformer.h"

#include <chrono>
#include <ctime>
#include <unistd.h>

#include <thread>

__global__
void interpolateChannels(const hipfftComplex* inputBuffer, hipfftComplex* summedSignals, const int i, const int* a, const int* b, const float* alpha, const float* beta)
{
    int id;    
    int l1 = blockIdx.x * blockDim.x + threadIdx.x; // internal index of this thread
    int l2 = blockIdx.x * blockDim.x + threadIdx.x + i * BLOCK_LEN; // global index of this thread

    // l1 -> 0 - 2047
    // l2 -> 0 - 2047 + i * 2048, i -> 0 - 168

    summedSignals[l2].x = 0.0f;
    for (int k = 0; k < NUM_CHANNELS; ++k)
    {
        id = k + i * NUM_CHANNELS;        
        if (max(0, -a[id]) == 0 && l1 < BLOCK_LEN - a[id]) // a >= 0            
            summedSignals[l2].x += alpha[id] * inputBuffer[l1 + a[id] + k * BLOCK_LEN].x; // do not write to the a[id] end positions
        else if (max(0, -a[id]) > 0 && l1 >= a[id]) 
            summedSignals[l2].x += alpha[id] * inputBuffer[l1 + a[id] + k * BLOCK_LEN].x; // do not write to the first a[id]-1 positions

        if (max(0, -b[id]) == 0 && l1 < BLOCK_LEN - b[id]) // b >= 0
            summedSignals[l2].x += beta[id] * inputBuffer[l1 + b[id] + k * BLOCK_LEN].x; // do not write to the b[id] end positions
        else if (max(0, -b[id]) > 0 && l1 >= b[id]) 
            summedSignals[l2].x += beta[id] * inputBuffer[l1 + b[id] + k * BLOCK_LEN].x; // do not write to the first b[id]-1 positions*/
    }    
}

__global__ 
void beamforming(const hipfftComplex* inputBuffer, const int* a, const int* b, float* alpha, const float* beta, hipfftComplex* summedSignals)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= NUM_BEAMS * NUM_BEAMS){
        return;
    }

    // interpolate channels    
    interpolateChannels<<<(BLOCK_LEN+255)/256, 256>>>(inputBuffer, summedSignals, i, a, b, alpha, beta);
    hipDeviceSynchronize();    
}

__global__
void bandpass_filtering_calcs(int i, hipfftComplex* summedSignals_fft_BP, hipfftComplex* summedSignals_fft, hipfftComplex* BP_filter)
{
    int l1 = blockIdx.x * blockDim.x + threadIdx.x; // internal index, from 0 to BLOCK_LEN - 1
    int l2 = blockIdx.x * blockDim.x + threadIdx.x + i * BLOCK_LEN; // internal index + compensation for which beam is being calced
    int l3 = blockIdx.x * blockDim.x + threadIdx.x + i * BLOCK_LEN * NUM_FILTERS; // as l2, but compensates for beams being calced in different freq-bands
    //       -           0 - 2047               -, + i *  2048     *     6

    for (int j = 0; j < NUM_FILTERS; ++j)
    {        
        summedSignals_fft_BP[l3 + j * BLOCK_LEN].x = summedSignals_fft[l2].x * BP_filter[l1 + j * BLOCK_LEN].x - summedSignals_fft[l2].y * BP_filter[l1 + j * BLOCK_LEN].y;
        summedSignals_fft_BP[l3 + j * BLOCK_LEN].y = summedSignals_fft[l2].x * BP_filter[l1 + j * BLOCK_LEN].y + summedSignals_fft[l2].y * BP_filter[l1 + j * BLOCK_LEN].x;        
    }
    // after these calculations there should be NUM_FILTERS signals per view, and each signals is BLOCK_LEN samples long, the strength of the signals need to be calced
}

__global__
void bandpass_filtering(hipfftComplex* summedSignals_fft_BP, hipfftComplex* summedSignals_fft, hipfftComplex* BP_filter, float* beams)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;    

    if (i >= NUM_BEAMS * NUM_BEAMS){
        return;
    }

    // calculations
    bandpass_filtering_calcs<<<(BLOCK_LEN+255)/256, 256>>>(i, summedSignals_fft_BP, summedSignals_fft, BP_filter);
    hipDeviceSynchronize();

    float beamstrength;
    int id;
    for (int j = 0; j < NUM_FILTERS; ++j)
    {
        beamstrength = 0.0f;
        for (int k = 0; k < BLOCK_LEN; ++k)
        {
            id = k + j * NUM_FILTERS + i * NUM_FILTERS * BLOCK_LEN;
            beamstrength += summedSignals_fft_BP[id].x * summedSignals_fft_BP[id].x + summedSignals_fft_BP[id].y * summedSignals_fft_BP[id].y;
        }
        beams[i + j * NUM_BEAMS * NUM_BEAMS] = 20 * log10(sqrtf(beamstrength) / ( (float)NUM_CHANNELS * (float)(BLOCK_LEN * BLOCK_LEN * sqrtf((float)BLOCK_LEN))));
    }
}


void free_resources(beamformingData* data)
{
    // free allocated memory
    free(data->beams);
    fftwf_free(data->ordbuffer);
    fftwf_free(data->temp);
    fftwf_free(data->block);
    hipFree(data->summedSignals);   
    hipFree(data->summedSignals_fft);
    hipFree(data->summedSignals_fft_BP);
    hipFree(data->BP_filter);
    hipFree(data->a);
    hipFree(data->alpha);
    hipFree(data->b);
    hipFree(data->beta);
    hipFree(data->gpu_block);
    hipFree(data->gpu_beams);
    fftwf_free(data->fft_data);    
    fftwf_free(data->filtered_data);    
    fftwf_free(data->LP_filter);        
    
    for (int i = 0; i < NUM_CHANNELS; ++i)
    {
        fftwf_destroy_plan(data->forw_plans[i]);
        fftwf_destroy_plan(data->back_plans[i]);                
    }

    hipfftDestroy(data->planMany);
    
    free(data->testsignal);
    free(data);
}

// Checks the return value of a PortAudio function. Logs the message and exits
// if there was an error
/*static void checkErr(PaError err, beamformingData* data) {
    if (err != paNoError) {
        printf("PortAudio error: %s\n", Pa_GetErrorText(err));
        free_resources(data);
        exit(EXIT_FAILURE);
    }
}*/

// PortAudio stream callback function. Will be called after every
// BLOCK_LEN audio samples PortAudio captures. Used to process the
// resulting audio sample.
/*static int streamCallback(
    const void* inputBuffer, void* outputBuffer, unsigned long framesPerBuffer,
    const PaStreamCallbackTimeInfo* timeInfo, PaStreamCallbackFlags statusFlags,
    void* userData
)*/
static void callBack(float* inputBuffer, beamformingData* data)
{
    // Cast our input buffer to a float pointer (since our sample format is `paFloat32`)
    float* in = (float*)inputBuffer;

    // We will not be modifying the output buffer. This line is a no-op.
    //(void)outputBuffer;

    //beamformingData* data = (beamformingData*)userData;
    
    // keep track of when to stop listening
    /*int finished;
    unsigned long framesLeft = data->maxFrameIndex - data->frameIndex;

    if( framesLeft < framesPerBuffer )
    {
        data->frameIndex += framesLeft;
        finished = paComplete;
    }
    else
    {
        data->frameIndex += framesPerBuffer;
        finished = paContinue;
    }*/

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    
    for (int i = 0; i < NUM_CHANNELS; ++i) // sort the incoming buffer based on channel
    {       
        for (int j = 0; j < BLOCK_LEN; ++j)
        {            
            //data->ordbuffer[i * BLOCK_LEN + j][0] = in[j * NUM_CHANNELS + i];
            data->ordbuffer[i * BLOCK_LEN + j][0] = in[j];
            data->ordbuffer[i * BLOCK_LEN + j][1] = 0.0f;            
        }        
    }

    for (int i = 0; i < NUM_CHANNELS; ++i) // build data block to be processed
    {
        // 1. move the last part of the old input into the beginning of the block
        // 2. fill the rest of the block with BLOCK_LEN - TEMP values from the new input
        // 3. save the last TEMP values from the new input to the temp storage for use in next call
        std::memcpy(&(data->block[i * BLOCK_LEN]), &(data->temp[i * TEMP]), TEMP * sizeof(fftwf_complex)); 
        std::memcpy(&(data->block[i * BLOCK_LEN + TEMP]), &(data->ordbuffer[i * BLOCK_LEN]), (BLOCK_LEN - TEMP) * sizeof(fftwf_complex));
        std::memcpy(&(data->temp[i * TEMP]), &(data->ordbuffer[i * BLOCK_LEN + (BLOCK_LEN - TEMP)]), TEMP * sizeof(fftwf_complex));
    }

    for (int i = 0; i < NUM_CHANNELS; ++i) // calculate fft for each channel
    {
        fftwf_execute(data->forw_plans[i]);
    }

    // perform lowpass filtering in freq domain
    int resultID, dataID;
    for (int i = 0; i < NUM_CHANNELS; ++i) // for every channel
    {
        for (int j = 0; j < FFT_OUTPUT_SIZE; ++j) // for all samples
        {                
            // j denotes frequency bin            
            // i denotes the channel
            resultID = j + i * FFT_OUTPUT_SIZE;
            dataID = j + i * FFT_OUTPUT_SIZE;
            data->filtered_data[resultID][0] = data->fft_data[dataID][0] * data->LP_filter[j][0] - data->fft_data[dataID][1] * data->LP_filter[j][1];
            data->filtered_data[resultID][1] = data->fft_data[dataID][0] * data->LP_filter[j][1] + data->fft_data[dataID][1] * data->LP_filter[j][0];                
        }
        // inverse fourier transform to get back signals in time domain.        
        fftwf_execute(data->back_plans[i]); // amplitude gain BLOCK_LEN
    }

    // copy data blocks to gpu
    hipMemcpy(data->gpu_block, data->block, BLOCK_LEN*NUM_CHANNELS*sizeof(fftwf_complex), hipMemcpyHostToDevice); // copy buffer to GPU memory    

    // create beams    
    beamforming<<<data->numBlocks, data->threadsPerBlock>>>(data->gpu_block, data->a, data->b, data->alpha, data->beta, data->summedSignals);
    hipDeviceSynchronize();

    hipMemcpy(data->testsignal, data->summedSignals, NUM_BEAMS*NUM_BEAMS*BLOCK_LEN*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    hipfftExecC2C(data->planMany, data->summedSignals, data->summedSignals_fft, HIPFFT_FORWARD);
    hipDeviceSynchronize();    
    
    bandpass_filtering<<<data->numBlocks, data->threadsPerBlock>>>(data->summedSignals_fft_BP, data->summedSignals_fft, data->BP_filter, data->gpu_beams);
    hipDeviceSynchronize();    

    // copy the intensity of the beams to the cpu
    hipMemcpy(data->beams, data->gpu_beams, NUM_BEAMS*NUM_BEAMS*NUM_FILTERS*sizeof(float), hipMemcpyDeviceToHost);

    /*float max;
    int maxid = 0;
    for (int f = 0; f < 1; ++f)
    {    
        printf("filter %d --------------------\n", f+1);
        for (int i = 0; i < NUM_BEAMS * NUM_BEAMS; ++i)
        {
            printf("%d: %f \n", i, data->beams[i + f * NUM_BEAMS * NUM_BEAMS]);
            if (i == 0)
                max = data->beams[i + f * NUM_BEAMS * NUM_BEAMS];
            else if(max < data->beams[i + f * NUM_BEAMS * NUM_BEAMS]){
                maxid = i;
                max = data->beams[i + f * NUM_BEAMS * NUM_BEAMS];
            }
        }
    }

    printf("max id: %d\n", maxid);
    printf("max: %f\n", max);*/

    end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed = end-start;

    std::cout << "elapsed: " << elapsed.count() << "s\n";

    //return finished;
}

int main() 
{
    // Initialize PortAudio
    /*PaError err;
    err = Pa_Initialize();
    checkErr(err, nullptr);

    // --------------------------------------------------------------------------------------------------------------
    // ------------------------ List all available audio devices and look for desired device ------------------------
    // --------------------------------------------------------------------------------------------------------------
    int numDevices = Pa_GetDeviceCount();
    printf("Number of devices: %d\n", numDevices);

    if (numDevices < 0){
        printf("Error getting device count.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }
    else if (numDevices == 0){
        printf("There are no available audio devices on this machine.\n");
        Pa_Terminate();        
        exit(EXIT_FAILURE);
    }

    int device = -1;
    const PaDeviceInfo* deviceInfo;
    for (int i = 0; i < numDevices; i++)
    {
        deviceInfo = Pa_GetDeviceInfo(i);
        printf("Device %d:\n", i);
        printf("    name: %s\n", deviceInfo->name);
        printf("    maxInputChannels: %d\n", deviceInfo->maxInputChannels);
        printf("    maxOutputChannels: %d\n", deviceInfo->maxOutputChannels);
        printf("    defaultSampleRate: %f\n", deviceInfo->defaultSampleRate);

        if (strcmp(deviceInfo->name, DEVICE_NAME) == 0)
        {
            device = i;
        }
    }

    if (device == -1){
        printf("\nDevice: %s not found!\n", DEVICE_NAME);
        Pa_Terminate();
        exit(EXIT_FAILURE);
    }

    printf("Device = %d\n", device);*/
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------
    // --------------------------------------------------------------------------------------------------------------    

    // setup necessary data containers for the beamforming
    beamformingData* data = (beamformingData*)malloc(sizeof(beamformingData));
    data->maxFrameIndex = NUM_SECONDS * SAMPLE_RATE; // Record for a few seconds.
    data->frameIndex = 0;

    if (NUM_BEAMS * NUM_BEAMS > MAX_THREADS_PER_BLOCK){
        data->numBlocks = (NUM_BEAMS * NUM_BEAMS) % MAX_THREADS_PER_BLOCK + 1;
        data->threadsPerBlock = dim3(MAX_THREADS_PER_BLOCK);
    }
    else{
        data->numBlocks = 1;
        data->threadsPerBlock = dim3(NUM_BEAMS * NUM_BEAMS);
    }

    printf("Setting up fir filters.\n");    
    py::scoped_interpreter python;

    py::function my_func =
        py::reinterpret_borrow<py::function>(
            py::module::import("filtercreation").attr("filtercreation")  
    );    
    
    py::list res = my_func(NUM_FILTERS, NUM_TAPS, BANDWIDTH); // create the filters
    // temporary save state of data
    std::vector<float> taps;
    for (py::handle obj : res) {  // iterators!
        taps.push_back(obj.attr("__float__")().cast<float>());
    }

    py::list res2 = my_func(1, NUM_TAPS, 15000.0f / 22050.0f);
    // temporary save state of data
    std::vector<float> taps2;
    for (py::handle obj : res2) {  // iterators!
        taps2.push_back(obj.attr("__float__")().cast<float>());
    }

    // transfer data for real, goal is to get a buffer that looks like (with zero-padded signals):
    // filter1[0], filter1[1], ..., 0, 0, 0, filter2[0], filter2[1], ..., 0, 0, 0
    // -------- BLOCK_LEN samples ---------, -------- BLOCK_LEN samples --------- 
    fftwf_complex* firfilters = (fftwf_complex*)malloc(BLOCK_LEN * NUM_FILTERS * sizeof(fftwf_complex));
    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        for (int j = 0; j < BLOCK_LEN; ++j)
        {
            if (j < NUM_TAPS)
                firfilters[i * BLOCK_LEN + j][0] = taps[NUM_TAPS * i + j];
            else
                firfilters[i * BLOCK_LEN + j][0] = 0.0f; // zero pad filters
            firfilters[i * BLOCK_LEN + j][1] = 0.0f;
        }
    }
    taps.clear();

    fftwf_complex* lpfilter = (fftwf_complex*)malloc(BLOCK_LEN * sizeof(fftwf_complex));
    for (int i = 0; i < BLOCK_LEN; ++i)
    {
        if (i < NUM_TAPS)
            lpfilter[i][0] = taps2[i];            
        else
            lpfilter[i][0] = 0.0f; // zero pad filters            
        lpfilter[i][1] = 0.0f;
    }
    taps2.clear();    

    // apply fft to filters
    fftwf_complex* firfiltersfft = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_FILTERS * sizeof(fftwf_complex));
    data->LP_filter = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * sizeof(fftwf_complex));
    fftwf_plan filter_plans[NUM_FILTERS];
    fftwf_plan lp_filter_plan;
    for (int i = 0; i < NUM_FILTERS; ++i) // create the plans for calculating the fft of each filter block
    {
        filter_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &firfilters[i * BLOCK_LEN], &firfiltersfft[i * FFT_OUTPUT_SIZE], FFTW_FORWARD, FFTW_ESTIMATE);
    }
    lp_filter_plan = fftwf_plan_dft_1d(BLOCK_LEN, lpfilter, data->LP_filter, FFTW_FORWARD, FFTW_ESTIMATE);

    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        fftwf_execute(filter_plans[i]);
    }
    fftwf_execute(lp_filter_plan);
    
    for (int i = 0; i < NUM_FILTERS; ++i)
    {
        fftwf_destroy_plan(filter_plans[i]);
    }
    fftwf_destroy_plan(lp_filter_plan);

    hipMalloc(&(data->BP_filter), sizeof(hipfftComplex) * BLOCK_LEN * NUM_FILTERS);
    hipMemcpy(data->BP_filter, firfiltersfft, sizeof(hipfftComplex) * BLOCK_LEN * NUM_FILTERS, hipMemcpyHostToDevice);
    //hipMemcpy(firfiltersfft, data->BP_filter, sizeof(hipfftComplex) * BLOCK_LEN * NUM_FILTERS, hipMemcpyDeviceToHost);

    std::vector<float> bins(BLOCK_LEN), f1(BLOCK_LEN), f2(BLOCK_LEN), f3(BLOCK_LEN), f4(BLOCK_LEN), f5(BLOCK_LEN), f6(BLOCK_LEN);
    
    for (int i = 0; i < BLOCK_LEN; ++i)
    {
        bins.at(i) = i;
        
        f1.at(i) = sqrt(firfiltersfft[i][0] * firfiltersfft[i][0] + firfiltersfft[i][1] * firfiltersfft[i][1]);
        f2.at(i) = sqrt(firfiltersfft[i + BLOCK_LEN][0] * firfiltersfft[i + BLOCK_LEN][0] + firfiltersfft[i + BLOCK_LEN][1] * firfiltersfft[i + BLOCK_LEN][1]);
        f3.at(i) = sqrt(firfiltersfft[i + 2 * BLOCK_LEN][0] * firfiltersfft[i + 2 * BLOCK_LEN][0] + firfiltersfft[i + 2 * BLOCK_LEN][1] * firfiltersfft[i + 2 * BLOCK_LEN][1]);
        f4.at(i) = sqrt(firfiltersfft[i + 3 * BLOCK_LEN][0] * firfiltersfft[i + 3 * BLOCK_LEN][0] + firfiltersfft[i + 3 * BLOCK_LEN][1] * firfiltersfft[i + 3 * BLOCK_LEN][1]);
        f5.at(i) = sqrt(firfiltersfft[i + 4 * BLOCK_LEN][0] * firfiltersfft[i + 4 * BLOCK_LEN][0] + firfiltersfft[i + 4 * BLOCK_LEN][1] * firfiltersfft[i + 4 * BLOCK_LEN][1]);
        f6.at(i) = sqrt(firfiltersfft[i + 5 * BLOCK_LEN][0] * firfiltersfft[i + 5 * BLOCK_LEN][0] + firfiltersfft[i + 5 * BLOCK_LEN][1] * firfiltersfft[i + 5 * BLOCK_LEN][1]);
    }

    /*plt::figure(1);
    plt::clf();    
    plt::plot(bins, f1);
    plt::xlabel("freq bin");
    plt::pause(0.25);

    plt::figure(2);
    plt::clf();    
    plt::plot(bins, f2);
    plt::xlabel("freq bin");

    plt::figure(3);
    plt::clf();    
    plt::plot(bins, f3);
    plt::xlabel("freq bin");

    plt::figure(4);
    plt::clf();    
    plt::plot(bins, f4);
    plt::xlabel("freq bin");
    plt::pause(0.25);

    plt::figure(5);
    plt::clf();    
    plt::plot(bins, f5);
    plt::xlabel("freq bin");

    plt::figure(6);
    plt::clf();    
    plt::plot(bins, f6);
    plt::xlabel("freq bin");*/

    free(firfilters);
    free(firfiltersfft);
    free(lpfilter);    

    printf("Create interpolation data.\n");
    float* theta = linspace(MIN_VIEW, NUM_BEAMS);
    float* phi = linspace(MIN_VIEW, NUM_BEAMS);
    float* delay = calcDelays(theta, phi);

    int* a = calca(delay);
    int* b = calcb(a);
    float* alpha = calcalpha(delay, b);
    float* beta = calcbeta(alpha);

    hipMalloc(&(data->a), sizeof(int) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->b), sizeof(int) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->alpha), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMalloc(&(data->beta), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_CHANNELS);
    hipMemcpy(data->a, a, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->b, b, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data->alpha, alpha, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data->beta, beta, NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float), hipMemcpyHostToDevice);
    free(theta); free(phi); free(delay); free(a); free(b); free(alpha); free(beta); // free memory which does not have to be allocated anymore*/    

    printf("Create remaining buffers\n");
    data->beams = (float*)malloc(NUM_BEAMS * NUM_BEAMS * NUM_FILTERS * sizeof(float));
    std::memset(data->beams, 0.0f, NUM_BEAMS * NUM_BEAMS * NUM_FILTERS * sizeof(float));
    hipMalloc(&(data->gpu_beams), sizeof(float) * NUM_BEAMS * NUM_BEAMS * NUM_FILTERS);

    hipMalloc(&(data->gpu_block), sizeof(hipfftComplex) * NUM_CHANNELS * BLOCK_LEN);

    data->temp = (fftwf_complex*)fftwf_malloc(TEMP * NUM_CHANNELS * sizeof(fftwf_complex));
    for (int i = 0; i < TEMP * NUM_CHANNELS; ++i)
    {
        data->temp[i][0] = 0.0f;
        data->temp[i][1] = 0.0f;
    }

    data->ordbuffer = (fftwf_complex*)fftwf_malloc(BLOCK_LEN * NUM_CHANNELS * sizeof(fftwf_complex));
    data->block = (fftwf_complex*)fftwf_malloc(BLOCK_LEN * NUM_CHANNELS * sizeof(fftwf_complex));
    
    hipMalloc(&(data->summedSignals), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN);
    hipMalloc(&(data->summedSignals_fft), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN);
    hipMalloc(&(data->summedSignals_fft_BP), sizeof(hipfftComplex) * NUM_BEAMS * NUM_BEAMS * BLOCK_LEN * NUM_FILTERS);

    data->fft_data = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_CHANNELS * sizeof(fftwf_complex));
    data->filtered_data = (fftwf_complex*)fftwf_malloc(FFT_OUTPUT_SIZE * NUM_CHANNELS * sizeof(fftwf_complex));    

    data->testsignal = (fftwf_complex*)fftwf_malloc(NUM_BEAMS * NUM_BEAMS * BLOCK_LEN * sizeof(fftwf_complex));    

    for (int i = 0; i < NUM_BEAMS * NUM_BEAMS * BLOCK_LEN; ++i)
    {
        data->testsignal[i][0] = 2.0f;
        data->testsignal[i][1] = 1.0f;
    }

    for (int i = 0; i < BLOCK_LEN * NUM_CHANNELS; ++i)
    {
        data->ordbuffer[i][0] = 0.0f;
        data->ordbuffer[i][1] = 0.0f;
    }
    
    for (int i = 0; i < BLOCK_LEN * NUM_CHANNELS; ++i)
    {
        data->block[i][0] = 0.0f;
        data->block[i][1] = 0.0f;
    }

    printf("Creating fft plans.\n");
    int n[1] = {BLOCK_LEN};
    int inembed[] = {BLOCK_LEN};
    int onembed[] = {BLOCK_LEN};
    
    hipfftPlanMany(&(data->planMany), 1, n, inembed, 1, BLOCK_LEN, onembed, 1, BLOCK_LEN, HIPFFT_C2C, NUM_BEAMS*NUM_BEAMS);

    for (int i = 0; i < NUM_CHANNELS; ++i) // create the plans for calculating the fft of each channel block
    {
        data->forw_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &data->block[i * BLOCK_LEN], &data->fft_data[i * FFT_OUTPUT_SIZE], FFTW_FORWARD, FFTW_ESTIMATE); // NUM_CHANNELS channels for each block which requires FFT_OUTPUT_SIZE spots to store the fft data
        data->back_plans[i] = fftwf_plan_dft_1d(BLOCK_LEN, &data->filtered_data[i * FFT_OUTPUT_SIZE], &data->block[i * BLOCK_LEN], FFTW_BACKWARD, FFTW_ESTIMATE);
    }

    float* input = (float*)malloc(BLOCK_LEN * 2 * sizeof(float));
    for (int i = 0; i < BLOCK_LEN * 2; ++i)
    {
        input[i] = cosf(2 * M_PI * 520.0f * (1.0f / SAMPLE_RATE) * i);// + cosf(2 * M_PI * 1700.0f * (1.0f / SAMPLE_RATE) * i) + \
                    cosf(2 * M_PI * 2750.0f * (1.0f / SAMPLE_RATE) * i) + cosf(2 * M_PI * 3400.0f * (1.0f / SAMPLE_RATE) * i);        
    }    

    // run the callback function 8 times    
    callBack(input, data);
    callBack(&(input[BLOCK_LEN]), data);
    
    printf("Done with callback.\n");

    free(input);

    /*std::vector<float> d(BLOCK_LEN), in(BLOCK_LEN), LP(BLOCK_LEN), filt(BLOCK_LEN);

    for (int i = 0; i < BLOCK_LEN; ++i)
    {
        in.at(i) = sqrt(data->fft_data[i][0] * data->fft_data[i][0] + data->fft_data[i][1] * data->fft_data[i][1]);
        LP.at(i) = sqrt(data->LP_filter[i][0] * data->LP_filter[i][0] + data->LP_filter[i][1] * data->LP_filter[i][1]);
        filt.at(i) = sqrt(data->filtered_data[i][0] * data->filtered_data[i][0] + data->filtered_data[i][1] * data->filtered_data[i][1]);
    }*/

    /*plt::figure(10);
    plt::title("Frequency contents, channel 1");
    plt::clf();    
    plt::plot(bins, in);
    plt::xlabel("freq bin");

    plt::figure(11);
    plt::title("Frequency contents, channel 1");
    plt::clf();    
    plt::plot(bins, LP);
    plt::xlabel("freq bin");

    plt::figure(12);
    plt::title("Frequency contents, channel 1");
    plt::clf();    
    plt::plot(bins, filt);
    plt::xlabel("freq bin");

    plt::show();*/
    
    /*printf("Defining stream parameters.\n");
    PaStreamParameters inputParameters;
    memset(&inputParameters, 0, sizeof(inputParameters));
    inputParameters.channelCount = NUM_CHANNELS;
    inputParameters.device = device;
    inputParameters.hostApiSpecificStreamInfo = NULL;
    inputParameters.sampleFormat = paFloat32;
    inputParameters.suggestedLatency = Pa_GetDeviceInfo(device)->defaultLowInputLatency;

    // Open the PortAudio stream
    printf("Starting stream.\n");    
    PaStream* stream;
    err = Pa_OpenStream(
        &stream,
        &inputParameters,
        NULL,
        SAMPLE_RATE,
        BLOCK_LEN,
        paNoFlag,
        streamCallback,
        data
    );
    checkErr(err, data);

    // Begin capturing audio
    err = Pa_StartStream(stream);
    checkErr(err, data);*/

    //FILE* signal = popen("gnuplot", "w");
    //FILE* signal2 = popen("gnuplot", "w");
    //FILE* signal3 = popen("gnuplot", "w");
    
    std::vector<float> d(BLOCK_LEN), fft_data(BLOCK_LEN), LP(BLOCK_LEN), block(BLOCK_LEN), summedsignal(BLOCK_LEN);
    //while( ( err = Pa_IsStreamActive( stream ) ) == 1 )    
    //{
        int beam = 84;
        for (int i = 0; i < BLOCK_LEN; ++i)
        {            
            fft_data.at(i) = sqrt(data->fft_data[i][0] * data->fft_data[i][0] + data->fft_data[i][1] * data->fft_data[i][1]);
            LP.at(i) = sqrt(data->LP_filter[i][0] * data->LP_filter[i][0] + data->LP_filter[i][1] * data->LP_filter[i][1]);
            block.at(i) = data->block[i][0];
            summedsignal.at(i) = data->testsignal[i + beam * BLOCK_LEN][0];
        }

        /*plt::figure(10);
        plt::clf();
        plt::plot(bins, fft_data);
        plt::xlabel("freq bin");
        plt::pause(0.25);

        plt::figure(11);
        plt::clf();
        plt::plot(bins, LP);
        plt::xlabel("freq bin");*/

        plt::figure(12);
        plt::clf();
        plt::plot(bins, block);
        plt::xlabel("time bin");

        plt::figure(13);
        plt::clf();
        plt::plot(bins, summedsignal);
        plt::xlabel("time bin");

        plt::show();

        //Pa_Sleep(250);
        // plot maximum direction
        /*plt::figure(1);
        plt::title("Max direction plot");
        plt::clf();
        plt::scatter(std::vector<float>{theta[data->thetaID] * 180.0f / (float)M_PI}, std::vector<float>{phi[data->phiID] * 180.0f / (float)M_PI}, 25.0, {{"color", "red"}});
        plt::xlim(MIN_VIEW, MAX_VIEW);
        plt::ylim(MIN_VIEW, MAX_VIEW);
        plt::xlabel("theta");
        plt::ylabel("phi");
        plt::grid(true);
        plt::pause(0.15);*/
        //printf("theta = %f\n", data->theta );
        //printf("phi = %f\n", data->phi );
        //printf("maxframeindex = %d\n", data->maxFrameIndex );
        //printf("frameindex = %d\n", data->frameIndex );
        //fflush(stdout);

        // plot frequency contents of channels
        /*plt::figure(2);
        plt::title("Frequency contents");
        plt::clf();
        for(int w = 0; w < NUM_CHANNELS; ++w){
            row = w / 4;
            col = w % 4;
            plt::subplot2grid(nrows, ncols, row, col);
            plt::plot({1.0,2.0,3.0,4.0});
            plt::xlabel("freq bin");
        }
        //plt::show();
        plt::pause(0.02);*/

        // plot beamforming results in color map
        /*fprintf(signal, "unset key\n");
        fprintf(signal, "set pm3d\n");
        fprintf(signal, "set view map\n");
        fprintf(signal, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal, "plot '-' matrix with image\n");
        
        for(int i = 0; i < NUM_BEAMS * NUM_BEAMS; i++) // plot map for the lowest frequency band    
        {
            fprintf(signal, "%f ", data->beams[i]);            
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal, "\n");            
        }
        
        fprintf(signal, "\ne\n");        
        fflush(signal);

        fprintf(signal2, "unset key\n");
        fprintf(signal2, "set pm3d\n");
        fprintf(signal2, "set view map\n");
        fprintf(signal2, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal2, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal2, "plot '-' matrix with image\n");
        
        for(int i = NUM_BEAMS * NUM_BEAMS; i < 2 * NUM_BEAMS * NUM_BEAMS; i++)
        {
            fprintf(signal2, "%f ", data->beams[i]);            
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal2, "\n");            
        }
        
        fprintf(signal2, "\ne\n");        
        fflush(signal2);*/

        /*fprintf(signal3, "unset key\n");
        fprintf(signal3, "set pm3d\n");
        fprintf(signal3, "set view map\n");
        fprintf(signal3, "set xrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal3, "set yrange [ -0.5 : %f ] \n", NUM_BEAMS - 0.5f);
        fprintf(signal3, "plot '-' matrix with image\n");
        
        for(int i = 0 * NUM_BEAMS * NUM_BEAMS; i < 1 * NUM_BEAMS * NUM_BEAMS; i++)
        {
            fprintf(signal3, "%f ", data->beams[i]);            
            if ((i+1) % NUM_BEAMS == 0)
                fprintf(signal3, "\n");            
        }
        
        fprintf(signal3, "\ne\n");        
        fflush(signal3);

        // Display the buffered changes to stdout in the terminal
        fflush(stdout);

        //plt::show();*/
    //}    

    // Stop capturing audio
    /*err = Pa_StopStream(stream);
    checkErr(err, data);

    // Close the PortAudio stream
    err = Pa_CloseStream(stream);
    checkErr(err, data);

    // Terminate PortAudio
    err = Pa_Terminate();
    checkErr(err, data);*/

    free_resources(data);
    return 0;
}

/////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////
/////////////////// UTILITY FUNCTIONS ///////////////////////
/////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////

float* linspace(int a, int num)
{
    // create a vector of length num
    //std::vector<double> v(NUM_BEAMS, 0);    
    float* f = (float*)malloc(NUM_BEAMS*sizeof(float));    
             
    // now assign the values to the array
    for (int i = 0; i < num; i++)
    {
        f[i] = (a + i * VIEW_INTERVAL) * M_PI / 180.0f;
    }
    return f;
}

float* calcDelays(float* theta, float* phi)
{
    float* d = (float*)malloc(NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float));    

    int pid = 0; // phi index
    int tid = 0; // theta index
    for (int i = 0; i < NUM_BEAMS * NUM_BEAMS; ++i){        
        for (int k = 0; k < NUM_CHANNELS; ++k){
            d[k + i * NUM_CHANNELS] = -(ya[k] * sinf(theta[tid]) * cosf(phi[pid]) + za[k] * sinf(phi[pid])) * ARRAY_DIST / C * SAMPLE_RATE;
        }
        tid++;
        if (tid >= NUM_BEAMS){
            tid = 0;
            pid++;
        }
    }
    return d;
}

int* calca(float* delay)
{
    int* a = (int*)malloc(NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int));
    for (int i = 0; i < NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS; ++i)
    {
        a[i] = floor(delay[i]);
    }
    return a;
}

int* calcb(int* a)
{
    int* b = (int*)malloc(NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(int));
    for (int i = 0; i < NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS; ++i)
    {
        b[i] = a[i] + 1;
    }
    return b;
}

float* calcalpha(float* delay, int* b)
{
    float* alpha = (float*)malloc(NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float));
    for (int i = 0; i < NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS; ++i)
    {
        alpha[i] = b[i] - delay[i];
    }
    return alpha;
}

float* calcbeta(float* alpha)
{
    float* beta = (float*)malloc(NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS*sizeof(float));
    for (int i = 0; i < NUM_BEAMS*NUM_BEAMS*NUM_CHANNELS; ++i)
    {
        beta[i] = 1 - alpha[i];
    }
    return beta;
}