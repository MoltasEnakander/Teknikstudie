
#include <hip/hip_runtime.h>
#include <stdio.h>

/*__global__
void saxpy(int n, float a, float* x, float* y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
	int N = 2048;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	cudaMalloc(&d_x, N*sizeof(float));
	cudaMalloc(&d_y, N*sizeof(float));

	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

	saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

	cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	{
		maxError = max(maxError, abs(y[i] - 4.0f));
	}
	printf("Max error: %f\n", maxError);

	cudaFree(d_x);
	cudaFree(d_y);
	free(x);
	free(y);
}*/



__global__ void child_k(const int i, const int j, const int FRAMES, const int NUM_VIEWS, float* summedSignals)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x + (i + j * NUM_VIEWS) * FRAMES;
	summedSignals[id] = id;
}

__global__ void parent_k(const int FRAMES, const int NUM_VIEWS, float* summedSignals)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	child_k<<<(FRAMES+255)/256, 256>>>(i, j, FRAMES, NUM_VIEWS, summedSignals);
}

int main(void)
{
	const int NUM_VIEWS = 13;
	const int FRAMES = 512;
	float* summedSignals = (float*)malloc(sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES); // each beam will have its own signal buffer of length FRAMES
	float* d_summedSignals;
	hipMalloc(&d_summedSignals, sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES);

	for (int i = 0; i < NUM_VIEWS * NUM_VIEWS * FRAMES; ++i)
	{
		summedSignals[i] = 0.0f;
	}

	hipMemcpy(d_summedSignals, summedSignals, sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES, hipMemcpyHostToDevice);

	int numBlocks = 1;
    dim3 threadsPerBlock(NUM_VIEWS, NUM_VIEWS);
	parent_k<<<numBlocks, threadsPerBlock>>>(FRAMES, NUM_VIEWS, d_summedSignals);

	hipMemcpy(summedSignals, d_summedSignals, sizeof(float) * NUM_VIEWS * NUM_VIEWS * FRAMES, hipMemcpyDeviceToHost);

	float error = 0.0f;
	for (int i = 0; i < NUM_VIEWS * NUM_VIEWS * FRAMES; ++i)
	{
		error += summedSignals[i] - i;
	}

	printf("error: %f\n", error);

	hipFree(d_summedSignals);
	free(summedSignals);
}